
#include <hip/hip_runtime.h>
#include<cstdio>

extern "C" {
__global__
void kernelMain(int *iTab, int *jTab, const unsigned int *arg){
    //int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    const unsigned int n=arg[0],m=arg[1];
    const unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int j=blockIdx.y*blockDim.y+threadIdx.y;
    const int rowSize=m;

    if(i>=n || j>=m)
        return;

    int a=i+1,b=j+1;
    int x=0,lastX=1;
    int y=1,lastY=0;
    while(b!=0)
    {
        int q=a/b;
        int c=a%b;
        a=b;
        b=c;
        c=x;
        x=lastX-q*x;
        lastX=c;
        c=y;
        y=lastY-q*y;
        lastY=c;
    }

    iTab[rowSize*i+j]=lastX;
    jTab[rowSize*i+j]=lastY;
}

}
