
#include <hip/hip_runtime.h>
#include<cstdio>

extern "C" {
__global__
void kernelMain(int *iTab, int*jTab){
    //int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int i=blockIdx.x,j=blockIdx.y;
    const int rowSize=gridDim.y;

    int a=i+1,b=j+1;
    int x=0,lastX=1;
    int y=1,lastY=0;
    while(b!=0)
    {
        int q=a/b;
        int c=a%b;
        a=b;
        b=c;
        c=x;
        x=lastX-q*x;
        lastX=c;
        c=y;
        y=lastY-q*y;
        lastY=c;
    }

    iTab[rowSize*i+j]=lastX;
    jTab[rowSize*i+j]=lastY;
}

}
