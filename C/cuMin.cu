
#include <hip/hip_runtime.h>
#include <cstdio>
#include <climits>

extern "C" {
__global__
void kernelMain(int *input, int *output){
    int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    __shared__ int mem[1024];
    mem[threadIdx.x]=input[thid];
    __syncthreads();

    for(unsigned int shift=1;shift<1024;shift*=2)
    {
        int val=mem[threadIdx.x];
        if(threadIdx.x>=shift)
        {
            if(val>mem[threadIdx.x-shift])
                val=mem[threadIdx.x-shift];
        }
        __syncthreads();
        mem[threadIdx.x]=val;
    }
    if(threadIdx.x==1023)
        output[blockIdx.x]=mem[1023];
}

__global__ void kernelPrepare(int *input, int *output, int* args)
{
    const unsigned int count=args[0];
    const unsigned int n=args[1];
    int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(thid*count>=n)
        return;
    int m=input[thid*count];
    for(unsigned int i=1;i<count && thid*count+i<n;++i)
    {
        if(m>input[thid*count+i])
            m=input[thid*count+i];
    }
    output[thid]=m;
}

}

