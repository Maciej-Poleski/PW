
#include <hip/hip_runtime.h>
#include<cstdio>

extern "C" {
__global__
void kernelMain(int *input, int *output, const unsigned int *arg){
    //int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    const unsigned int width=arg[0],height=arg[1];
    const unsigned int column=blockIdx.x*32+threadIdx.x;
    const unsigned int row=blockIdx.y*32+threadIdx.y;

    const unsigned int newRow=(32*blockIdx.x+threadIdx.y);
    const unsigned int newColumn=32*blockIdx.y+threadIdx.x;

    __shared__ int cache[32][35];

    cache[threadIdx.x][threadIdx.y]=input[width*row+column];
    __syncthreads();
    output[newRow*height+newColumn]=cache[threadIdx.y][threadIdx.x];
//     output[height*column+row]=cache[threadIdx.y][threadIdx.x];
//     output[height*column+row]=input[width*row+column];
}

}

