
#include <hip/hip_runtime.h>
#include <cstdio>
#include <climits>

#define SERIAL_SCALE 2

#define SERIAL_PART (1<<SERIAL_SCALE)

extern "C" {
__global__
void kernelMain(int *input, int *output){
    int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    __shared__ int mem[1024];
    int m=input[thid*SERIAL_PART];
    for(unsigned int i=1;i<SERIAL_PART;++i)
    {
        int t=input[thid*SERIAL_PART+i];
        if(t<m)
            m=t;
    }
    mem[threadIdx.x]=m;
    __syncthreads();

    for(unsigned int shift=1;shift<1024;shift*=2)
    {
        int val=mem[threadIdx.x];
        if(threadIdx.x>=shift)
        {
            if(val>mem[threadIdx.x-shift])
                val=mem[threadIdx.x-shift];
        }
        __syncthreads();
        mem[threadIdx.x]=val;
    }
    if(threadIdx.x==1023)
        output[blockIdx.x]=mem[1023];
}

__global__ void kernelPrepare(int *input, int *output, int* args)
{
    const unsigned int count=args[0];
    const unsigned int n=args[1];
    int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(thid*count>=n)
        return;
    int m=input[thid*count];
    for(unsigned int i=1;i<count && thid*count+i<n;++i)
    {
        if(m>input[thid*count+i])
            m=input[thid*count+i];
    }
    output[thid]=m;
}

}

