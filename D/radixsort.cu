
#include <hip/hip_runtime.h>
#include <cstdio>
#include <climits>

#define SERIAL_SCALE 1

#define SERIAL_PART (1<<SERIAL_SCALE)

extern "C" {
    /**
     * @param input Cała tablica do posortowania
     * @param output Wynik: Wszystkie prefix sum z kolejnych bloków
     * @param args args[0]=pozycja bitu (0..30)
     */
__global__
void kernelMain(int *input, int *output, int *sumGlobal, int bitMask){
    //int thid = (blockIdx.x * blockDim.x) + threadIdx.x; // Globalny ID wątku (mało przydatne)
    __shared__ int mem[SERIAL_PART][1024+3];
    int baseOffset=(blockIdx.x * blockDim.x)*SERIAL_PART; // Przesunięcie obecnego ciągu bloków

    for(int i=0;i<SERIAL_PART;++i)
    {
        mem[i][threadIdx.x]= ((input[baseOffset+i*blockDim.x+threadIdx.x]&bitMask)==0);
    }
    __syncthreads();
    for(int shift=1;shift<1024;shift*=2)
    {
        //scan
        for(int i=0;i<SERIAL_PART;++i)
        {
            int v=mem[i][threadIdx.x];
            if(threadIdx.x>=shift)
            {
                v+=mem[i][threadIdx.x-shift];
            }
            __syncthreads();
            mem[i][threadIdx.x]=v;
        }
    }
    //__syncthreads();
    for(int i=0;i<SERIAL_PART;++i)
    {
        output[baseOffset+i*blockDim.x+threadIdx.x]=mem[i][threadIdx.x];
        if(threadIdx.x == blockDim.x-1)
        {
            sumGlobal[blockIdx.x * SERIAL_PART + i + 1] = mem[i][threadIdx.x];
        }
    }
}

/**
 * @param input Wejściowa tablica liczb
 * @param output Wyjściowa tablica liczb
 * @param sumLocal Tablica prefix sum kolejnych bloków
 * @param sumGlobal Tablica prefix sum (zewnętrzna) ostatnich sum z kolejnych bloków
 * @param args args[0]=pozycja bitu (0..30), args[1]=pozycja pierwszej liczby z 1 w output
 * @param
 */
__global__
void kernelShuffle(int *input,int *output,int *sumLocal,int *sumGlobal,int bitMask,int sumLast)
{
    int baseOffset=(blockIdx.x * blockDim.x)*SERIAL_PART; // Przesunięcie obecnego ciągu bloków
    __shared__ int sumGlobalS[SERIAL_PART][1024];
        for(int i=0;i<SERIAL_PART;++i)
        {
	    int in=baseOffset+i*blockDim.x+threadIdx.x;
            sumGlobalS[i][threadIdx.x]=sumGlobal[blockIdx.x*SERIAL_PART+i]+sumLocal[in];
        }
    __syncthreads();
    for(int i=0;i<SERIAL_PART;++i)
    {
        int in=baseOffset+i*blockDim.x+threadIdx.x;
        if((input[in]&bitMask)==0)
        {
            //printf("%d -> [%d]\n",input[in],sumLocal[in]+sumGlobal[blockIdx.x*SERIAL_PART+i]-1);
            output[sumGlobalS[i][threadIdx.x]-1]=input[in];
        }
        else
        {
            //printf("%d -> [%d]\n",input[in],args[1]+in-(sumLocal[in]+sumGlobal[blockIdx.x*SERIAL_PART+i]));
            output[sumLast+in-(sumGlobalS[i][threadIdx.x])]=input[in];
        }
    }
}

}

