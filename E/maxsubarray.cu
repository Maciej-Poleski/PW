
#include <hip/hip_runtime.h>
#include <cstdio>
#include <climits>
#include <algorithm>

#define SERIAL_SCALE 3

#define SERIAL_PART (1<<SERIAL_SCALE)

extern "C" {
    __global__
    void kernelMain(int* input, int* output, int N)
    {
        //int thid = (blockIdx.x * blockDim.x) + threadIdx.x; // Globalny ID wątku (mało przydatne)
        __shared__ int mem[SERIAL_PART * 1024];
        for (int i = 0; i < SERIAL_PART; ++i) {
            mem[blockDim.x * i + threadIdx.x] = input[N * blockIdx.y + blockDim.x * i + threadIdx.x];
        }
        __syncthreads();
        for (int shift = 1; shift < N; shift *= 2) {
            int v[SERIAL_PART];
            for (int i = 0; i < SERIAL_PART; ++i) {
                v[i] = mem[blockDim.x * i + threadIdx.x];
                if (shift <= blockDim.x * i + threadIdx.x)
                    v[i] += mem[blockDim.x * i + threadIdx.x - shift];
            }
            __syncthreads();
            for (int i = 0; i < SERIAL_PART; ++i) {
                mem[blockDim.x * i + threadIdx.x] = v[i];
            }
        }

        for (int i = 0; i < SERIAL_PART; ++i) {
            output[(N + 1)*blockIdx.y + blockDim.x * i + threadIdx.x + 1] = mem[blockDim.x * i + threadIdx.x];
        }

        if (threadIdx.x == 0)
            output[(N + 1)*blockIdx.y] = 0;
    }

    __global__
    void findMax(int* prefixSumMatrix, int* output, int N)
    {
        int x = (blockIdx.x * blockDim.x) + threadIdx.x;
        int y = (blockIdx.y * blockDim.y) + threadIdx.y + 1;
        __shared__ int mem[1024];
        int flatId = blockDim.x * threadIdx.y + threadIdx.x;
        if (x >= y) {
            mem[flatId] = 0;
        } else {
            int result = 0;
            int current = 0;
            for (int i = 0; i < N; ++i) {
                current = current + prefixSumMatrix[(N + 1) * i + y] - prefixSumMatrix[(N + 1) * i + x];
                if (current < 0)
                    current = 0;
                if (current > result)
                    result = current;
            }
            mem[flatId] = result;
        }
        __syncthreads();
        for (int shift = 1; shift < 1024; shift *= 2) {
            int v = mem[flatId];
            if (flatId >= shift)
                if (v < mem[flatId - shift])
                    v = mem[flatId - shift];
            __syncthreads();
            mem[flatId] = v;
        }
        if (flatId == 1023) {
            output[gridDim.x * blockIdx.y + blockIdx.x] = mem[flatId];
        }
    }
}

