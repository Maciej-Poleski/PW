
#include <hip/hip_runtime.h>
#include <cstdio>
#include <climits>
#include <algorithm>

#define SERIAL_SCALE 3

#define SERIAL_PART (1<<SERIAL_SCALE)

extern "C" {
    __global__
    void initialize(int* output, int size)
    {
        int flatId=(blockIdx.x * blockDim.x) + threadIdx.x;
        if(flatId<size)
            output[flatId]=flatId;
    }

    __global__
    void transpose(int *input, int *output, int N){
        const unsigned int column=blockIdx.x*32+threadIdx.x;
        const unsigned int row=blockIdx.y*32+threadIdx.y;

        const unsigned int newRow=(32*blockIdx.x+threadIdx.y);
        const unsigned int newColumn=32*blockIdx.y+threadIdx.x;

        __shared__ int cache[32][35];

        cache[threadIdx.x][threadIdx.y]=input[N*row+column];
        __syncthreads();
        output[newRow*N+newColumn]=cache[threadIdx.y][threadIdx.x];
    }

    __global__
    void propagateMin(int *G,int *result,int *modified,int N)
    {
        int column=(blockIdx.x * blockDim.x) + threadIdx.x;
        bool m=false;
#define GET(array,row) array[N*(row)+column]
#define propagate(i)                                                           \
        int currG=GET(G,i);                                                    \
        if(prevG==currG)                                                       \
        {                                                                      \
            int currR=GET(result,i);                                           \
            if(currR>prevR)                                                    \
            {                                                                  \
                GET(result,i)=prevR;                                           \
                m=true;                                                        \
            }                                                                  \
            else                                                               \
            {                                                                  \
                prevR=currR;                                                   \
            }                                                                  \
        }                                                                      \
        else                                                                   \
        {                                                                      \
            prevR=GET(result,i);                                               \
        }                                                                      \
        prevG=currG;

        int prevG=GET(G,0);
        int prevR=GET(result,0);
        for(int i=1;i<N;++i)
        {
            propagate(i)
        }

        prevG=GET(G,N-1);
        prevR=GET(result,N-1);
        for(int i=N-2;i>=0;--i)
        {
            propagate(i)
        }

        if(m)
            *modified=-1;

#undef propagate
#undef GET
    }
}

