
#include <hip/hip_runtime.h>
#include<cstdio>

extern "C" {
    __global__
    void kernelMain(int *input, int *output, const unsigned int *arg) {
        //int thid = (blockIdx.x * blockDim.x) + threadIdx.x;

        __shared__ int cache[32][32];

        const unsigned int width=arg[0],height=arg[1];

        const unsigned int x=threadIdx.x;


        for(unsigned int y=0; y<32; ++y)
        {
            const unsigned int row=blockIdx.y*32+y;
            const unsigned int column=blockIdx.x*32+x;

            cache[x][y]=input[width*row+column];
        }
        __syncthreads();
        for(unsigned int y=0; y<32; ++y)
        {
            const unsigned int newRow=32*blockIdx.x+x;
            const unsigned int newColumn=32*blockIdx.y+y;


            output[newRow*height+newColumn]=cache[x][y];
        }
    }

}

